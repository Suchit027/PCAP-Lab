#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void reverse(char *string, int n){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n && (string[i] == ' ' || i == 0)){
        if (i == 0){
            i -= 1;
        }
        int j = i + 1;
        while (string[j] != '\0' && string[j] != ' '){
            j += 1;
        }
        j -= 1;
        if (j - i > 0){
            i += 1;
            for(int k = 0; k < (j - i) / 2; ++k){
                char t = string[i + k];
                string[i + k] = string[j - k];
                string[j - k] = t;
            }
        }
    }
    return;
}

int main(){
    char *string, *dstring;
    int n;
    printf("enter the string length\n");
    scanf("%d", &n);
    getchar();
    string = (char *)malloc((n + 1) * sizeof(char));
    printf("enter the string\n");
    fgets(string, n + 1, stdin);
    hipMalloc((void **)&dstring, (n + 1) * sizeof(char));
    hipMemcpy(dstring, string, (n + 1) * sizeof(char), hipMemcpyHostToDevice);
    reverse<<<1, n>>>(dstring, n);
    hipMemcpy(string, dstring, (n + 1) * sizeof(char), hipMemcpyDeviceToHost);
    printf("answer - %s", string);
    return 0;
}