#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void newstring(char *inp, char *ans, int l){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < l){
        int copy = l - i;
        int j = ((l * (l + 1)) / 2) - ((copy * (copy + 1)) / 2);
        for(int k = 0; k < copy; ++k){
            ans[j + k] = inp[k];
        }
    }
    return;
}

int main(){
    char *arr, *ans;
    int l;
    printf("enter word length\n");
    scanf("%d", &l);
    printf("enter word\n");
    arr = (char *)malloc(sizeof(char) * (l + 1));
    scanf("%s", arr);
    ans = (char *)malloc(sizeof(char) * (((l * (l + 1)) / 2) + 1));
    char *darr, *dans;
    hipMalloc((void **)&darr, sizeof(char) * (l + 1));
    hipMalloc((void **)&dans, sizeof(char) * (((l * (l + 1)) / 2) + 1));
    hipMemcpy(darr, arr, sizeof(char) * (l + 1), hipMemcpyHostToDevice);
    newstring<<<1, 100>>>(darr, dans, l);
    hipMemcpy(ans, dans, sizeof(char) * (((l * (l + 1)) / 2) + 1), hipMemcpyDeviceToHost);
    hipFree(darr);
    hipFree(dans);
    printf("%s", ans);
    return 0;
}