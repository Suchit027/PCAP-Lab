#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void create(char *input, char *ans, int n, int l){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if(i < l){
        char c = input[i];
        for(int j = 0; j < n; ++j){
            ans[i + (l * j)] = c;
        }
    }
    return;
}

int main(){
    char *input, *dinput, *ans, *dans;
    int n, l;
    printf("enter length of string\n");
    scanf("%d", &l);
    getchar();
    input = (char *)malloc((l + 1) * sizeof(char));
    printf("enter n value\n");
    scanf("%d", &n);
    getchar();
    ans = (char *)malloc(((l * n) + 1) * sizeof(char));
    printf("enter string\n");
    fgets(input, l + 1, stdin);
    hipMalloc((void **)&dinput, (l + 1) * sizeof(char));
    hipMalloc((void **)&dans, ((l * n) + 1) * sizeof(char));
    hipMemcpy(dinput, input, (l + 1) * sizeof(char), hipMemcpyHostToDevice);
    create<<<1, l>>>(dinput, dans, n, l);
    hipMemcpy(ans, dans, ((n * l) + 1) * sizeof(char), hipMemcpyDeviceToHost);
    ans[(n * l)] = '\0';
    printf("answer - %s", ans);
    hipFree(dans);
    hipFree(dinput);
    return 0;
}