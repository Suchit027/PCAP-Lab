#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"


__global__ void find(char *arr, int la, char *word, int lw, int *ans) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i + lw <= la) {
        int found = 1;
        for (int j = 0; j < lw; ++j) {
            if (word[j] != arr[i + j]) {
                found = 0;
                break;
            }
        }
        if (found == 1) {
            atomicAdd(ans, 1);
        }
    }
}

int main() {
    int la, lw, ans = 0, *dans;
    char *arr, *word, *darr, *dword;

    printf("Enter string length\n");
    scanf("%d", &la);
    getchar();

    arr = (char *)malloc(sizeof(char) * (la + 1));
    
    printf("Enter string\n");
    fgets(arr, la + 1, stdin);

    printf("Enter word size\n");
    scanf("%d", &lw);
    getchar();

    word = (char *)malloc(sizeof(char) * (lw + 1));
    
    printf("Enter word\n");
    fgets(word, lw + 1, stdin);

    hipMalloc((void **)&darr, sizeof(char) * (la + 1));
    hipMalloc((void **)&dword, sizeof(char) * (lw + 1));
    hipMalloc((void **)&dans, sizeof(int));

    hipMemcpy(darr, arr, sizeof(char) * (la + 1), hipMemcpyHostToDevice);
    hipMemcpy(dword, word, sizeof(char) * (lw + 1), hipMemcpyHostToDevice);
    hipMemset(dans, 0, sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (la + threadsPerBlock - 1) / threadsPerBlock;

    find<<<blocksPerGrid, threadsPerBlock>>>(darr, la, dword, lw, dans);

    hipMemcpy(&ans, dans, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dans);
    hipFree(darr);
    hipFree(dword);

    printf("Occurrences: %d\n", ans);

    free(arr);
    free(word);

    return 0;
}
