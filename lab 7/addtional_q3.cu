#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void create(char *input, int n, char *ans){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n){
        int j = (i * (i + 1)) / 2;
        for (int k = 0; k < i + 1; ++k){
            ans[j + k] = input[i];
        }
    }
    return;
}

int main(){
    char *input, *dinput, *ans, *dans;
    int n;
    printf("enter length of string\n");
    scanf("%d", &n);
    getchar();
    printf("enter string\n");
    input = (char *)malloc((n + 1) * sizeof(char));
    fgets(input, n + 1, stdin);
    ans = (char *)malloc((((n * (n + 1)) / 2) + 1) * sizeof(char));
    hipMalloc((void **)&dinput, (n + 1) * sizeof(char));
    hipMalloc((void **)&dans, (((n * (n + 1)) / 2) + 1) * sizeof(char));
    hipMemcpy(dinput, input, (n + 1) * sizeof(char), hipMemcpyHostToDevice);
    create<<<1, n>>>(dinput, n, dans);
    hipMemcpy(ans, dans, (((n * (n + 1)) / 2) + 1) * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(dinput);
    hipFree(dans);
    printf("answer - %s", ans);
    return 0;
}