#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void diagonal(int *a){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    a[(i * blockDim.x) + i] = 0;
    return;
}

__global__ void factorial(int *a){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    for(int j = i + 1; j < blockDim.x; ++j){
        int add = a[(i * blockDim.x) + j];
        int val = 1;
        while(add > 0){
            val *= add;
            add -= 1;
        }
        a[(i * blockDim.x) + j] = val;
    }
    return;
}

__global__ void summ(int *a){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    for(int j = 0; j < i; ++j){
        int val = a[(i * blockDim.x) + j];
        int add = 0;
        while(val > 0){
            add += val % 10;
            val /= 10;
        }
        a[(i * blockDim.x) + j] = add;
    }
    return;
}

int main(){
    int *a, n, *da;
    printf("enter n\n");
    scanf("%d", &n);
    a = (int *)malloc(sizeof(int) * n * n);
    printf("enter a\n");
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, n * n * sizeof(int));
    hipMemcpy(da, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    diagonal<<<1, n>>>(da);
    factorial<<<1, n>>>(da);
    summ<<<1, n>>>(da);
    hipMemcpy(a, da, n * n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", a[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    return 0;
}