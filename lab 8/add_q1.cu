#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void rowsum(int *a, int *ans, int n){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    ans[id] = 0;
    for(int i = 0; i < n; ++i){
        ans[id] += a[(id * n) + i];
    }
    return;
}

__global__ void colsum(int *a, int *ans, int m, int n){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    ans[id] = 0;
    for(int i = 0; i < m; ++i){
        ans[id] += a[(i * n) + id];
    }
    return;
}

__global__ void solve(int *a, int *row, int *col){
    int r = blockIdx.x;
    int c = threadIdx.x;
    if(a[(r * blockDim.x) + c] % 2 == 0){
        a[(r * blockDim.x) + c] = row[r];
    }
    else{
        a[(r * blockDim.x) + c] = col[c];
    }
    return;
}

int main(){
    int *a, *drow, *dcol, *da, m, n;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(sizeof(int) * m * n);
    printf("enter a\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, m * n * sizeof(int));
    hipMalloc((void **)&drow, m * sizeof(int));
    hipMalloc((void **)&dcol, n * sizeof(int));
    hipMemcpy(da, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    rowsum<<<1, m>>>(da, drow, n);
    colsum<<<1, n>>>(da, dcol, m, n);
    solve<<<m, n>>>(da, drow, dcol);
    hipMemcpy(a, da, m * n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", a[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(drow);
    hipFree(dcol);
    return 0;
}