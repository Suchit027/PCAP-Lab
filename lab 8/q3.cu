#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int m, int n, int *ans){
    int r = blockIdx.x;
    int c = threadIdx.x;
    ans[(r * n) + c] = a[(r * n) + c] + b[(r * n) + c];
    return;
}

int main(){
    int *a, *b, *ans, m, n, *da, *db, *dans;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(m * n * sizeof(int));
    b = (int *)malloc(m * n * sizeof(int));
    ans = (int *)malloc(m * n * sizeof(int));
    printf("enter a\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    printf("enter b\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &b[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, m * n * sizeof(int));
    hipMalloc((void **)&db, m * n * sizeof(int));
    hipMalloc((void **)&dans, m * n * sizeof(int));
    hipMemcpy(da, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, m * n * sizeof(int), hipMemcpyHostToDevice);
    add<<<m, n>>>(da, db, m, n, dans);
    hipMemcpy(ans, dans, m * n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", ans[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(a);
    hipFree(b);
    hipFree(ans);
    return 0;
}