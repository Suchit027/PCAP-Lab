#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void mul(int *a, int *b, int *ans, int m1, int n1, int n2){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    for(int i = 0; i < m1; ++i){
        ans[(i * n2) + id] = 0;
        for(int j = 0; j < n1; ++j){
            ans[(i * n2) + id] += a[(i * n1) + j] * b[(j * n2) + id];
        }
    }
    return;
}

int main(){
    int *a, *b, *ans, *da, *db, *dans, m1, n1, m2, n2;
    printf("enter m1, n1, m2, n2\n");
    scanf("%d %d %d %d", &m1, &n1, &m2, &n2);
    a = (int *)malloc(sizeof(int) * m1 * n1);
    b = (int *)malloc(sizeof(int) * m2 * n2);
    ans = (int *)malloc(sizeof(int) * m1 * n2);
    printf("enter a\n");
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n1; ++j){
            scanf("%d", &a[(i * n1) + j]);
        }
    }
    printf("enter b\n");
    for(int i = 0; i < m2; ++i){
        for(int j = 0; j < n2; ++j){
            scanf("%d", &b[(i * n2) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m1 * n1);
    hipMalloc((void **)&db, sizeof(int) * m2 * n2);
    hipMalloc((void **)&dans, sizeof(int) * m1 * n2);
    hipMemcpy(da, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    mul<<<1, n2>>>(da, db, dans, m1, n1, n2);
    hipMemcpy(ans, dans, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n2; ++j){
            printf("%d ", ans[(i * m1) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dans);
    return 0;
}