#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int n, int *ans){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    for(int i = 0; i < n; ++i){
        ans[(id * n) + i] = 0;
        ans[(id * n) + i] = a[(id * n) + i] + b[(id * n) + i];
    }
    return;
}

int main(){
    int *a, *b, *ans, m, n, *da, *db, *dans;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(m * n * sizeof(int));
    b = (int *)malloc(m * n * sizeof(int));
    ans = (int *)malloc(m * n * sizeof(int));
    printf("enter a\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    printf("enter b\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &b[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, m * n * sizeof(int));
    hipMalloc((void **)&db, m * n * sizeof(int));
    hipMalloc((void **)&dans, m * n * sizeof(int));
    hipMemcpy(da, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, m * n * sizeof(int), hipMemcpyHostToDevice);
    add<<<1, m>>>(da, db, n, dans);
    hipMemcpy(ans, dans, m * n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", ans[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(a);
    hipFree(b);
    hipFree(ans);
    return 0;
}