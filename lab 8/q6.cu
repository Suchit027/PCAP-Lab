#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void mul(int *a, int *b, int *ans, int n1, int n2){
    int r = blockIdx.x;
    int c = threadIdx.x;
    ans[(r * n2) + c] = 0;
    for(int i = 0; i < n1; ++i){
        ans[(r * n2) + c] += a[(r * n1) + i] * b[(i * n2) + c];
    }
    return;
}

int main(){
    int *a, *b, *ans, *da, *db, *dans, m1, n1, m2, n2;
    printf("enter m1, n1, m2, n2\n");
    scanf("%d %d %d %d", &m1, &n1, &m2, &n2);
    a = (int *)malloc(sizeof(int) * m1 * n1);
    b = (int *)malloc(sizeof(int) * m2 * n2);
    ans = (int *)malloc(sizeof(int) * m1 * n2);
    printf("enter a\n");
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n1; ++j){
            scanf("%d", &a[(i * n1) + j]);
        }
    }
    printf("enter b\n");
    for(int i = 0; i < m2; ++i){
        for(int j = 0; j < n2; ++j){
            scanf("%d", &b[(i * n2) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m1 * n1);
    hipMalloc((void **)&db, sizeof(int) * m2 * n2);
    hipMalloc((void **)&dans, sizeof(int) * m1 * n2);
    hipMemcpy(da, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    mul<<<m1, n2>>>(da, db, dans, n1, n2);
    hipMemcpy(ans, dans, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n2; ++j){
            printf("%d ", ans[(i * m1) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dans);
    return 0;
}