#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void csr(int *data, int *col, int *row, int *b, int *ans){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    ans[i] = 0;
    for(int start = row[i]; start < row[i + 1]; ++start){
        ans[i] += data[start] * b[col[start]];
    }
    return;
}

int main(){
    int *a, *data, *col, *row, *ddata, *dcol, *drow, *b, *db, *ans, *dans, m, n;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    printf("enter a\n");
    a = (int *)malloc(sizeof(int) * m * n);
    int length = 0;
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
            if(a[(i * n) + j] != 0){
                length += 1;
            }
        }
    }
    data = (int *)malloc(sizeof(int) * length);
    col = (int *)malloc(sizeof(int) * length);
    row = (int *)malloc(sizeof(int) * (m + 1));
    int k = 0, x = 0;
    for(int i = 0; i < m; ++i){
        row[i] = x;
        for(int j = 0; j < n; ++j){
            if(a[(i * n) + j] != 0){
                data[k] = a[(i * n) + j];
                col[k++] = j;
                x += 1;
            }
        }
    }
    row[m] = length;
    printf("enter b\n");
    b = (int *)malloc(sizeof(int) * n);
    for(int i = 0; i < n; ++i){
        scanf("%d", &b[i]);
    }
    ans = (int *)malloc(sizeof(int) * m);
    hipMalloc((void **)&ddata, sizeof(int) * length);
    hipMalloc((void **)&dcol, sizeof(int) * length);
    hipMalloc((void **)&drow, sizeof(int) * (m + 1));
    hipMalloc((void **)&db, sizeof(int) * n);
    hipMalloc((void **)&dans, sizeof(int) * m);
    hipMemcpy(ddata, data, sizeof(int) * length, hipMemcpyHostToDevice);
    hipMemcpy(dcol, col, sizeof(int) * length, hipMemcpyHostToDevice);
    hipMemcpy(drow, row, sizeof(int) * (m + 1), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * n, hipMemcpyHostToDevice);
    csr<<<1, m>>>(ddata, dcol, drow, db, dans);
    hipMemcpy(ans, dans, sizeof(int) * m, hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        printf("%d ", ans[i]);
    }
    hipFree(db);
    hipFree(ddata);
    hipFree(dans);
    hipFree(drow);
    hipFree(dcol);
    return 0;
}