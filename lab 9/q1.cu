#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MyKernel(int *a, int *b, int *c, int N) {

    int x = blockIdx.y * blockDim.y + threadIdx.y;

    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y <N && x < N) {

        int sum = 0;

        for (int k = 0; k < N; k++) {

        sum += a[x * N + k] * b[x + k * N];

        }

        c[y + x * N ] = sum;

    }

}

int main() {
    int A[256];
    for (int i = 0; i < 256; i++)
        A[i] = 1;

    int B[256];
    for (int i = 0; i < 256; i++)
        B[i] = 2;

    int C[256];

    int size = sizeof(int) * 256;
    
    int *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    MyKernel<<<(8, 2), (1, 16)>>>(d_A, d_A, d_C, 16);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++){
        printf("%d ", C[i]);}
    return 0;
}
