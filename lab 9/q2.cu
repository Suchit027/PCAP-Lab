#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void change(int *a, int n){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    for(int j = 0; j < n; ++j){
        int val = a[(i * n) + j];
        for(int k = 1; k < i + 1; ++k){
            val *= a[(i * n) + j];
        }
        a[(i * n) + j] = val;
    }
    return;
}

int main(){
    int *a, m, n, *da;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(sizeof(int) * m * n);
    printf("enter a\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m * n);
    hipMemcpy(da, a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    change<<<1, m>>>(da, n);
    hipMemcpy(a, da, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", a[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    return 0;
}