#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void makestr(char *a, int *prefix, int n, char *ans)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    for (int k = prefix[(i * n) + j]; k < prefix[(i * n) + j + 1]; ++k)
    {
        ans[k] = a[(i * n) + j];
    }
    return;
}
int main()
{
    char *a, *ans, *da, *dans;
    int *prefix, m, n, *dprefix;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (char *)malloc(sizeof(char) * m * n);
    prefix = (int *)malloc(sizeof(int) * ((m * n) + 1));
    printf("enter a\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            getchar();
            scanf("%c", &a[(i * n) + j]);
        }
    }
    printf("enter b\n");
    prefix[0] = 0;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            int val = 0;
            scanf("%d", &val);
            prefix[(i * n) + j + 1] = prefix[(i * n) + j] + val;
        }
    }
    ans = (char *)malloc(sizeof(char) * (prefix[m * n] + 1));
    hipMalloc((void **)&da, sizeof(char) * m * n);
    hipMalloc((void **)&dans, sizeof(char) * prefix[m * n]);
    hipMalloc((void **)&dprefix, sizeof(int) * ((m * n) + 1));
    hipMemcpy(da, a, sizeof(char) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(dprefix, prefix, sizeof(int) * ((m * n) + 1), hipMemcpyHostToDevice);
    makestr<<<m, n>>>(da, dprefix, n, dans);
    hipMemcpy(ans, dans, sizeof(char) * prefix[m * n], hipMemcpyDeviceToHost);
    ans[prefix[m * n]] = '\0';
    printf("answer - %s", ans);
    return 0;
}