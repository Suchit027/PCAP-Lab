#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void change(int *a, int m, int n){
    int i = blockIdx.x;
    int j = threadIdx.x;
    if (i > 0 && i < m - 1 && j > 0 && j < n - 1){
        a[(i * n) + j] = ~a[(i * n) + j] & 0xF;
    }
    return;
}

int main(){
    int *a, m, n, *da;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(sizeof(int) * m * n);
    hipMalloc((void **)&da, sizeof(int) * m * n);
    printf("enter a\n");
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            scanf("%d", &a[(i * n) + j]);
        }
    }
    hipMemcpy(da, a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    change<<<m, n>>>(da, m, n);
    hipMemcpy(a, da, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < n; ++j){
            printf("%d ", a[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    return 0;
}