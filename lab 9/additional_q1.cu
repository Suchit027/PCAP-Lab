#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void rowsum(int *a, int n, int *row)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    row[i] = 0;
    for (int j = 0; j < n; ++j)
    {
        row[i] += a[(i * n) + j];
    }
    return;
}

__global__ void colsum(int *a, int n, int *col)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    col[i] = 0;
    for (int j = 0; j < n; ++j)
    {
        col[i] += a[(j * n) + i];
    }
    return;
}

__global__ void replace(int *a, int n, int *row, int *col)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    a[(i * n) + j] = row[i] + col[j];
    return;
}

int main()
{
    int *a, m, n, *da, *drow, *dcol;
    printf("enter m and n\n");
    scanf("%d %d", &m, &n);
    a = (int *)malloc(sizeof(int) * m * n);
    printf("enter a\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            scanf("%d", &a[(i * n) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m * n);
    hipMalloc((void **)&drow, sizeof(int) * m);
    hipMalloc((void **)&dcol, sizeof(int) * n);
    hipMemcpy(da, a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    rowsum<<<1, m>>>(da, n, drow);
    colsum<<<1, n>>>(da, n, dcol);
    replace<<<m, n>>>(da, n, drow, dcol);
    hipMemcpy(a, da, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", a[(i * n) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(drow);
    hipFree(dcol);
    return 0;
}