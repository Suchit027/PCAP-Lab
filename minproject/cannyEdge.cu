#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define BLOCK_SIZE 16

__global__ void gaussianBlurKernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        float kernel[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
        float sum = 0;
        float normalization = 16.0;
        float blurred_pixel = 0.0;

        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                blurred_pixel += input[(y + i) * width + (x + j)] * kernel[i + 1][j + 1];
                sum += kernel[i + 1][j + 1];
            }
        }

        output[idx] = (unsigned char)(blurred_pixel / normalization);
    }
}

__global__ void sobelFilterKernel(unsigned char *input, float *gradient, float *direction, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        float Gx = 0, Gy = 0;
        int idx = y * width + x;
        
        int sobelX[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
        int sobelY[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
        
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int pixel = input[(y + i) * width + (x + j)];
                Gx += pixel * sobelX[i + 1][j + 1];
                Gy += pixel * sobelY[i + 1][j + 1];
            }
        }
        
        gradient[idx] = sqrtf(Gx * Gx + Gy * Gy);
        direction[idx] = atan2f(Gy, Gx);
    }
}

__global__ void nonMaxSuppressionKernel(float *gradient, float *direction, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        float angle = direction[idx] * 180.0 / M_PI;
        angle = fmodf(angle + 180.0, 180.0);
        float grad = gradient[idx];

        int neighbor1, neighbor2;
        if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle < 180)) {
            neighbor1 = y * width + (x - 1);
            neighbor2 = y * width + (x + 1);
        } else if (angle >= 22.5 && angle < 67.5) {
            neighbor1 = (y - 1) * width + (x + 1);
            neighbor2 = (y + 1) * width + (x - 1);
        } else if (angle >= 67.5 && angle < 112.5) {
            neighbor1 = (y - 1) * width + x;
            neighbor2 = (y + 1) * width + x;
        } else {
            neighbor1 = (y - 1) * width + (x - 1);
            neighbor2 = (y + 1) * width + (x + 1);
        }

        if (grad >= gradient[neighbor1] && grad >= gradient[neighbor2]) {
            output[idx] = (unsigned char)grad;
        } else {
            output[idx] = 0;
        }
    }
}

__global__ void hysteresisKernel(unsigned char *input, unsigned char *output, int width, int height, int lowThreshold, int highThreshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        if (input[idx] >= highThreshold) {
            output[idx] = 255;
        } else if (input[idx] >= lowThreshold) {
            output[idx] = 128;
        } else {
            output[idx] = 0;
        }
    }
}

void cannyEdgeDetection(unsigned char *d_input, unsigned char *d_output, int width, int height) {
    unsigned char *d_blurred;
    float *d_gradient, *d_direction;
    hipMalloc(&d_blurred, width * height * sizeof(unsigned char));
    hipMalloc(&d_gradient, width * height * sizeof(float));
    hipMalloc(&d_direction, width * height * sizeof(float));
    
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    gaussianBlurKernel<<<gridSize, blockSize>>>(d_input, d_blurred, width, height);
    sobelFilterKernel<<<gridSize, blockSize>>>(d_blurred, d_gradient, d_direction, width, height);
    nonMaxSuppressionKernel<<<gridSize, blockSize>>>(d_gradient, d_direction, d_output, width, height);
    hysteresisKernel<<<gridSize, blockSize>>>(d_output, d_output, width, height, 50, 150);
    
    hipFree(d_blurred);
    hipFree(d_gradient);
    hipFree(d_direction);
}

int main() {
   // Load image using OpenCV
   Mat image = imread("input.jpg", IMREAD_GRAYSCALE);
   if (image.empty()) {
       printf("Error: Could not open image!\n");
       return -1;
   }
   int width = image.cols;
   int height = image.rows;
   
   // Allocate memory
   unsigned char *d_input, *d_output;
   hipMalloc(&d_input, width * height * sizeof(unsigned char));
   hipMalloc(&d_output, width * height * sizeof(unsigned char));
   
   // Copy data to GPU
   hipMemcpy(d_input, image.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
   
   // Perform Canny edge detection
   cannyEdgeDetection(d_input, d_output, width, height);
   
   // Copy result back to CPU
   Mat outputImage(height, width, CV_8UC1);
   hipMemcpy(outputImage.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
   
   // Save the result
   imwrite("output.jpg", outputImage);
   
   // Cleanup
   hipFree(d_input);
   hipFree(d_output);
   
   return 0;
}
