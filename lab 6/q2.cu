#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void sort(int *arr, int l_arr, int *ans){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if(i < l_arr){
        int val = arr[i];
        int idx = 0;
        for(int j = 0; j < l_arr; ++j){
            if (arr[j] < val || (arr[j] == val && j < i)){
                idx += 1;
            }
        }
        ans[idx] = val;
    }
    return;
}

int main(int argc, char **argv){
    int *arr, *ans, n;
    printf("enter size of array\n");
    scanf("%d", &n);
    arr = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for(int i = 0; i < n; ++i){
        scanf("%d", &arr[i]);
    }
    int *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));
    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    sort<<<1, n>>>(darr, n, dans);
    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dans);
    hipFree(darr);
    for(int i = 0; i < n; ++i){
        printf("%d ", ans[i]);
    }
    return 0;
}