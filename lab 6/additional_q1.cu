#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void octal(int *arr, int *ans, int n){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if(i < n){
        ans[i] = 0;
        while(arr[i] > 0){
            ans[i] = (arr[i] % 8) + (10 * ans[i]);
            arr[i] /= 8;
        }
        int val = 0;
        while(ans[i] > 0){
            val = (10 * val) + (ans[i] % 10);
            ans[i] /= 10;
        }
        ans[i] = val;
    }
    return;
}

int main(int argc, char **argv){
    int *arr, *ans, n;
    printf("enter size of array\n");
    scanf("%d", &n);
    arr = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for(int i = 0; i < n; ++i){
        scanf("%d", &arr[i]);
    }
    int *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));
    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    octal<<<1, n>>>(darr, dans, n);
    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dans);
    hipFree(darr);
    for(int i = 0; i < n; ++i){
        printf("%d ", ans[i]);
    }
    return 0;
}