#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void evenPhase(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    if (tid % 2 == 0){
        i = tid;
    }
    else{
        i = tid + 1;
    }

    if (i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

__global__ void oddPhase(int *arr, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    if (tid % 2 != 0){
        i = tid;
    }
    else{
        i = tid + 1;
    }
    if (i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

void oddEvenSort(int *arr, int n)
{
    int *d_arr;
    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    for (int i = 0; i < n; i++)
    {
        evenPhase<<<gridSize, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
        oddPhase<<<gridSize, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main(int argc, char **argv)
{
    int n = 5;
    int arr[] = {5, 3, 1, 2, 4};

    printf("Initial array: ");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", arr[i]);
    }

    oddEvenSort(arr, n);
    printf("\n");

    for (int i = 0; i < n; i++)
    {
        printf("%d ", arr[i]);
    }

    return 0;
}
