#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"


__global__ void comp(int *arr, int *ans, int n) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n) {
        ans[i] = ~arr[i];
    }
}

int binary_to_int(const char *binary_str) {
    int result = 0;
    for (int i = 0; binary_str[i] != '\0'; i++) {
        result = result * 2 + (binary_str[i] - '0');
    }
    return result;
}

void int_to_binary(int num, char *binary_str, int size) {
    for (int i = size - 1; i >= 0; i--) {
        binary_str[size - 1 - i] = (num & (1 << i)) ? '1' : '0';
    }
    binary_str[size] = '\0';
}

int main() {
    int *arr, *ans, n;
    printf("Enter size of array: ");
    scanf("%d", &n);

    arr = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);

    printf("Enter binary numbers (as strings of 1's and 0's):\n");
    for (int i = 0; i < n; ++i) {
        char binary_str[33];
        scanf("%s", binary_str);
        arr[i] = binary_to_int(binary_str);
    }

    int *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));

    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    comp<<<numBlocks, blockSize>>>(darr, dans, n);

    hipDeviceSynchronize();


    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(dans);
    hipFree(darr);


    printf("\nOne's complement in binary:\n");
    for (int i = 0; i < n; ++i) {
        char binary_result[33];
        int_to_binary(ans[i], binary_result, 32);
        printf("%s\n", binary_result);
    }

    free(arr);
    free(ans);

    return 0;
}
