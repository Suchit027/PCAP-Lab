#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void conv(int *arr, int *mask, int l_arr, int l_mask, int *ans){
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int val = 0;
    int start = i - (l_mask / 2);
    for(int j = 0; j < l_mask; ++j){
        if (j + start >= 0 && j + start < l_arr){
            val += arr[j + start] * mask[j];
        }
    }
    ans[i] = val;
}

int main(int argc, char **argv){
    printf("enter array size\n");
    int n;
    scanf("%d", &n);
    int *arr = (int *)malloc(n * sizeof(int));
    int *ans = (int *)malloc(n * sizeof(int));
    printf("enter array\n");
    for (int i = 0; i < n; ++i){
        scanf("%d", &arr[i]);
    }
    printf("enter mask size\n");
    int m;
    scanf("%d", &m);
    printf("enter mask\n");
    int *mask = (int *)malloc(m *sizeof(int));
    for(int i = 0; i < m; ++i){
        scanf("%d", &mask[i]);
    }
    int *dmask, *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));
    hipMalloc((void **)&dmask, m * sizeof(int));
    hipMemcpy(dmask, mask, m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    conv<<<1, n>>>(darr, dmask, n, m, dans);
    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < n; ++i){
        printf("%d ", ans[i]);
    }
    hipFree(dans);
    hipFree(dmask);
    hipFree(darr);
    return 0;
}