#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__constant__ int mask[5];

__global__ void conv1d(int *a, int *ans, int mask_l, int input_l)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int start = i - (mask_l / 2);
    ans[i] = 0;
    for (int j = 0; j < mask_l; ++j)
    {
        if (start + j >= 0 && start + j < input_l)
        {
            ans[i] += a[start + j] * mask[j];
        }
    }
    return;
}
int main()
{
    int *a, *ans, *da, *dans, *m, mask_l, input_l;
    printf("enter input size\n");
    scanf("%d", &input_l);
    a = (int *)malloc(sizeof(int) * input_l);
    ans = (int *)malloc(sizeof(int) * input_l);
    printf("enter a\n");
    for (int i = 0; i < input_l; ++i)
    {
        scanf("%d", &a[i]);
    }
    printf("enter mask size\n");
    scanf("%d", &mask_l);
    m = (int *)malloc(sizeof(int) * mask_l);
    printf("enter mask\n");
    for (int i = 0; i < mask_l; ++i)
    {
        scanf("%d", &m[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * input_l);
    hipMalloc((void **)&dans, sizeof(int) * input_l);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), m, sizeof(int) * mask_l);
    hipMemcpy(da, a, sizeof(int) * input_l, hipMemcpyHostToDevice);
    conv1d<<<1, input_l>>>(da, dans, mask_l, input_l);
    hipMemcpy(ans, dans, sizeof(int) * input_l, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < input_l; ++i)
    {
        printf("%d ", ans[i]);
    }
    hipFree(da);
    hipFree(dans);
    return 0;
}