#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_SIZE 5
#define MASK_RADIUS 2

__constant__ int mask[5];

__global__ void conv1d(int *a, int *ans, int input_l)
{
    __shared__ int m[TILE_SIZE + (2 * MASK_RADIUS)];
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < input_l)
    {
        m[threadIdx.x + MASK_RADIUS] = a[i];
        if (threadIdx.x < MASK_RADIUS)
        {
            if (i - MASK_RADIUS > 0)
            {
                m[threadIdx.x] = a[i - MASK_RADIUS];
            }
            else
            {
                m[threadIdx.x] = 0;
            }
        }
        if (threadIdx.x + (2 * MASK_RADIUS) >= TILE_SIZE)
        {
            if (i + MASK_RADIUS < input_l)
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = a[i + MASK_RADIUS];
            }
            else
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = 0;
            }
        }
        __syncthreads();
        ans[i] = 0;
        for (int j = -MASK_RADIUS; j < MASK_RADIUS + 1; ++j)
        {
            ans[i] += m[threadIdx.x + MASK_RADIUS + j] * mask[MASK_RADIUS + j];
        }
    }
    return;
}