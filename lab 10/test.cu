#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 2

__global__ void multiply(int *a, int *b, int *c, int a_n, int b_n, int a_m){
    __shared__ int m[TILE_WIDTH][TILE_WIDTH];
    __shared__ int n[TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int pval = 0;
    if(row >= a_m || col >= b_n){
        return;
    }
    int total_phases = (b_n + TILE_WIDTH - 1) / TILE_WIDTH;
    for(int phase = 0; phase < total_phases; ++phase){
        int a_col = (phase * TILE_WIDTH) + threadIdx.x;
        int b_row = (phase * TILE_WIDTH) + threadIdx.y;
        if(a_col >= a_n || b_row >= a_n){
            return;
        }
        m[threadIdx.y][threadIdx.x] = a[(row * a_n) + a_col];
        n[threadIdx.y][threadIdx.x] = b[(b_row * b_n) + col];
        __syncthreads();
        for(int k = 0; k < TILE_WIDTH; ++k){
            pval += m[threadIdx.y][k] * n[k][threadIdx.x];
        }
        __syncthreads();
    }
    c[(b_n * row) + col] = pval;
    return;
}

int main(){
    int *a, *b, *c, m1, m2, n1, n2, *da, *db, *dc;
    printf("enter m1 n1 m2 n2\n");
    scanf("%d %d %d %d", &m1, &n1, &m2, &n2);
    a = (int *)malloc(sizeof(int) * m1 * n1);
    b = (int *)malloc(sizeof(int) * m2 * n2);
    c = (int *)malloc(sizeof(int) * m1 * n2);
    printf("enter a\n");
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n1; ++j){
            scanf("%d", &a[(i * n1) + j]);
        }
    }
    printf("enter b\n");
    for(int i = 0; i < m2; ++i){
        for(int j = 0; j < n2; ++j){
            scanf("%d", &b[(i * n2) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m1 * n1);
    hipMalloc((void **)&db, sizeof(int) * m2 * n2);
    hipMalloc((void **)&dc, sizeof(int) * m1 * n2);
    hipMemcpy(da, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    dim3 gridSize((n2 + TILE_WIDTH - 1) / TILE_WIDTH, (m1 + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    multiply<<<gridSize, blockSize>>>(da, db, dc, n1, n2, m1);
    hipMemcpy(c, dc, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    printf("answer\n");
    for(int i = 0; i < m1; ++i){
        for(int j = 0; j < n2; ++j){
            printf("%d ", c[(i * n2) + j]);
        }
        printf("\n");
    }
    return 0;
}