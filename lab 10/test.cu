#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void evenphase(int *a, int input_l)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i % 2 != 0)
    {
        i += 1;
    }
    if (i < input_l - 1 && a[i] > a[i + 1])
    {
        int temp = a[i];
        a[i] = a[i + 1];
        a[i + 1] = temp;
    }
    return;
}
__global__ void oddphase(int *a, int input_l)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i % 2 == 0)
    {
        i += 1;
    }
    if (i < input_l - 1 && a[i] > a[i + 1])
    {
        int temp = a[i];
        a[i] = a[i + 1];
        a[i + 1] = temp;
    }
    return;
}
int main()
{
    int *a, n, *da;
    printf("enter size of array\n");
    scanf("%d", &n);
    a = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for (int i = 0; i < n; ++i)
    {
        scanf("%d", &a[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * n);
    hipMemcpy(da, a, sizeof(int) * n, hipMemcpyHostToDevice);
    for (int i = 0; i < n; ++i)
    {
        evenphase<<<1, n>>>(da, n);
        hipDeviceSynchronize();
        oddphase<<<1, n>>>(da, n);
        hipDeviceSynchronize();
    }
    hipMemcpy(a, da, sizeof(int) * n, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < n; ++i)
    {
        printf("%d ", a[i]);
    }
    hipFree(da);
    return 0;
}