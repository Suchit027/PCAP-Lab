#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_SIZE 5
#define MASK_RADIUS 2

__constant__ int mask[5];

__global__ void conv1d(int *a, int n, int *ans)
{
    __shared__ int m[TILE_SIZE + (2 * MASK_RADIUS)];
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        m[threadIdx.x + MASK_RADIUS] = a[i];
        if (threadIdx.x < MASK_RADIUS)
        {
            if (i - MASK_RADIUS < 0)
            {
                m[threadIdx.x] = 0;
            }
            else
            {
                m[threadIdx.x] = a[i - threadIdx.x];
            }
        }
        if (threadIdx.x + (2 * MASK_RADIUS) >= TILE_SIZE)
        {
            if (i + MASK_RADIUS < n)
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = a[i + threadIdx.x];
            }
            else
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = 0;
            }
        }
        __syncthreads();
        int val = 0;
        for (int j = -MASK_RADIUS; j < MASK_RADIUS + 1; ++j)
        {
            val += m[threadIdx.x + MASK_RADIUS + j] * mask[MASK_RADIUS + j];
        }
        ans[i] = val;
        __syncthreads();
    }
    return;
}
int main()
{
    int *a, *ans, n, *da, *dans, *mmask;
    printf("enter array size\n");
    scanf("%d", &n);
    a = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);
    mmask = (int *)malloc(sizeof(int) * 5);
    printf("enter array\n");
    for (int i = 0; i < n; ++i)
    {
        scanf("%d", &a[i]);
    }
    printf("enter mask\n");
    for (int i = 0; i < 5; ++i)
    {
        scanf("%d", &mmask[i]);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(mask), mmask, sizeof(int) * 5);
    hipMalloc((void **)&da, sizeof(int) * n);
    hipMalloc((void **)&dans, sizeof(int) * n);
    hipMemcpy(da, a, sizeof(int) * n, hipMemcpyHostToDevice);
    conv1d<<<1, n>>>(da, n, dans);
    hipMemcpy(ans, dans, sizeof(int) * n, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < n; ++i)
    {
        printf("%d", ans[i]);
    }
    return 0;
}