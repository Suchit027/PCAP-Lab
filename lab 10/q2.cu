#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__constant__ int mask[20];

__global__ void convconstant(int *a, int *b, int mask_len, int a_len){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int val = 0;
    int start = i - (mask_len / 2);
    for(int j = 0; j < mask_len; ++j){
        if(start + j >= 0 && start + j < a_len){
            val += a[start + j] * mask[j];
        }
    }
    b[i] = val;
    return;
}

int main(){
    int *a, *da, *b, *db, *maskk, n, m;
    printf("enter mask size\n");
    scanf("%d", &m);
    printf("enter mask\n");
    maskk = (int *)malloc(sizeof(int) * m);
    for(int i = 0; i < m; ++i){
        scanf("%d", &maskk[i]);
    }
    printf("enter array size\n");
    scanf("%d", &n);
    a = (int *)malloc(sizeof(int) * n);
    b = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for(int i = 0; i < n; ++i){
        scanf("%d", &a[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * n);
    hipMalloc((void **)&db, sizeof(int) * n);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), maskk, sizeof(int) * m);
    hipMemcpy(da, a, sizeof(int) * n, hipMemcpyHostToDevice);
    convconstant<<<1, n>>>(da, db, m, n);
    hipMemcpy(b, db, sizeof(int) * n, hipMemcpyDeviceToHost);
    printf("answer - \n");
    for(int i = 0; i < n; ++i){
        printf("%d ", b[i]);
    }
    hipFree(da);
    hipFree(db);
    return 0;
}