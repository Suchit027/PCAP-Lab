#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define MAXLEN 20 

__global__ void inclusive(int *a, int a_len){
    __shared__ int temp[MAXLEN];
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    temp[i] = a[i];
    __syncthreads();
    for(int stride = 1; stride < a_len; stride *= 2){
        int val = 0;
        if(stride <= i){
            val = temp[i - stride];
        }
        __syncthreads();
        temp[i] += val;
        __syncthreads();
    }
    a[i] = temp[i];
    return;
}

int main(){
    int *a, n, *da;
    printf("enter array size\n");
    scanf("%d", &n);
    a = (int *)malloc(sizeof(int) * n);
    printf("enter array\n");
    for(int i = 0; i < n; ++i){
        scanf("%d", &a[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * n);
    hipMemcpy(da, a, sizeof(int) * n, hipMemcpyHostToDevice);
    inclusive<<<1, n>>>(da, n);
    hipMemcpy(a, da, sizeof(int) * n, hipMemcpyDeviceToHost);
    printf("answer \n");
    for(int i = 0; i < n; ++i){
        printf("%d ", a[i]);
    } 
    hipFree(da);
    return 0;
}