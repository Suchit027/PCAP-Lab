#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define MASK_RADIUS 2
#define TILE_SIZE 8

__constant__ int mask[10];

__global__ void conv1d(int *a, int input_l, int *ans)
{
    __shared__ int m[TILE_SIZE + (2 * MASK_RADIUS) + 1];
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < input_l)
    {
        m[MASK_RADIUS + i] = a[i];
        if (threadIdx.x < MASK_RADIUS)
        {
            if (i - MASK_RADIUS >= 0)
            {
                m[i] = a[i - MASK_RADIUS];
            }

            else
            {
                m[i] = 0;
            }
        }
        if (threadIdx.x + MASK_RADIUS >= blockDim.x)
        {
            if (i + (2 * MASK_RADIUS) < input_l)
            {
                m[i + (2 * MASK_RADIUS)] = a[i + MASK_RADIUS];
            }

            else
            {
                m[i + (2 * MASK_RADIUS)] = 0;
            }
        }
        __syncthreads();
        for (int j = -MASK_RADIUS; j < MASK_RADIUS + 1; ++j)
        {
            ans[i] += m[i + MASK_RADIUS + j] * mask[MASK_RADIUS + j];
        }
    }
    return;
}

int main()
{
    int *a, *m, *ans, *da, *dans, input_l;
    printf("enter input length\n");
    scanf("%d", &input_l);
    a = (int *)malloc(sizeof(int) * input_l);
    ans = (int *)malloc(sizeof(int) * input_l);
    printf("enter a\n");
    for (int i = 0; i < input_l; ++i)
    {
        scanf("%d", &a[i]);
    }
    printf("enter mask\n");
    m = (int *)malloc(sizeof(int) * ((2 * MASK_RADIUS) + 1));
    for (int i = 0; i < (2 * MASK_RADIUS) + 1; ++i)
    {
        scanf("%d", &m[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * input_l);
    hipMalloc((void **)&dans, sizeof(int) * input_l);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), m, sizeof(int) * ((2 * MASK_RADIUS) + 1));
    hipMemcpy(da, a, sizeof(int) * input_l, hipMemcpyHostToDevice);
    dim3 gridSize((input_l + TILE_SIZE - 1) / TILE_SIZE);
    conv1d<<<gridSize, TILE_SIZE>>>(da, input_l, dans);
    hipMemcpy(ans, dans, sizeof(int) * input_l, hipMemcpyDeviceToHost);
    printf("answer - \n");
    for (int i = 0; i < input_l; ++i)
    {
        printf("%d ", ans[i]);
    }
    hipFree(da);
    hipFree(dans);
    return 0;
}