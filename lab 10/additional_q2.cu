#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define MASK_RADIUS 2
#define TILE_SIZE 5

__constant__ int mask[5];

__global__ void conv1d(int *a, int *ans, int input_l)
{
    __shared__ int m[TILE_SIZE + (2 * MASK_RADIUS)];
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < input_l)
    {
        m[MASK_RADIUS + threadIdx.x] = a[i];
        if (threadIdx.x < MASK_RADIUS)
        {
            if (i - MASK_RADIUS >= 0)
            {
                m[threadIdx.x] = a[i - MASK_RADIUS];
            }
            else
            {
                m[threadIdx.x] = 0;
            }
        }
        if (threadIdx.x + MASK_RADIUS >= TILE_SIZE)
        {
            if (i + MASK_RADIUS < input_l)
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = a[i + MASK_RADIUS];
            }
            else
            {
                m[threadIdx.x + (2 * MASK_RADIUS)] = 0;
            }
        }
        __syncthreads();
        ans[i] = 0;
        for (int j = -MASK_RADIUS; j < MASK_RADIUS + 1; ++j)
        {
            ans[i] += m[threadIdx.x + MASK_RADIUS + j] * mask[MASK_RADIUS + j];
        }
    }
    return;
}

int main()
{
    int *a, *m, *ans, *da, *dans, input_l;
    printf("enter input length\n");
    scanf("%d", &input_l);
    a = (int *)malloc(sizeof(int) * input_l);
    ans = (int *)malloc(sizeof(int) * input_l);
    printf("enter a\n");
    for (int i = 0; i < input_l; ++i)
    {
        scanf("%d", &a[i]);
    }
    printf("enter mask\n");
    m = (int *)malloc(sizeof(int) * ((2 * MASK_RADIUS) + 1));
    for (int i = 0; i < (2 * MASK_RADIUS) + 1; ++i)
    {
        scanf("%d", &m[i]);
    }
    hipMalloc((void **)&da, sizeof(int) * input_l);
    hipMalloc((void **)&dans, sizeof(int) * input_l);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), m, sizeof(int) * ((2 * MASK_RADIUS) + 1));
    hipMemcpy(da, a, sizeof(int) * input_l, hipMemcpyHostToDevice);
    dim3 gridSize((input_l + TILE_SIZE - 1) / TILE_SIZE);
    conv1d<<<gridSize, TILE_SIZE>>>(da, dans, input_l);
    hipMemcpy(ans, dans, sizeof(int) * input_l, hipMemcpyDeviceToHost);
    printf("answer - \n");
    for (int i = 0; i < input_l; ++i)
    {
        printf("%d ", ans[i]);
    }
    hipFree(da);
    hipFree(dans);
    return 0;
}