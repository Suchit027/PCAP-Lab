#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 2

__global__ void mul(int *a, int *b, int *c, int a_n, int b_n)
{
    __shared__ int md[TILE_WIDTH][TILE_WIDTH];
    __shared__ int nd[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int pval = 0;

    if (row >= gridDim.y * TILE_WIDTH || col >= b_n)
    {
        return;
    }

    int num_phases = (a_n + TILE_WIDTH - 1) / TILE_WIDTH;
    for (int phase = 0; phase < num_phases; ++phase)
    {
        int tiled_col = (phase * TILE_WIDTH) + threadIdx.x;
        int tiled_row = (phase * TILE_WIDTH) + threadIdx.y;

        if (tiled_col >= a_n || tiled_row >= a_n)
        {
            return;
        }

        md[threadIdx.y][threadIdx.x] = a[row * a_n + tiled_col];

        nd[threadIdx.y][threadIdx.x] = b[tiled_row * b_n + col];

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            pval += md[threadIdx.y][k] * nd[k][threadIdx.x];
        }

        __syncthreads();
    }

    c[row * b_n + col] = pval;
}

int main()
{
    int *a, *b, *c, m1, n1, m2, n2, *da, *db, *dc;
    printf("enter m1, n1, m2, n2\n");
    scanf("%d %d %d %d", &m1, &n1, &m2, &n2);
    a = (int *)malloc(sizeof(int) * m1 * n1);
    b = (int *)malloc(sizeof(int) * m2 * n2);
    c = (int *)malloc(sizeof(int) * m1 * n2);
    printf("enter a\n");
    for (int i = 0; i < m1; ++i)
    {
        for (int j = 0; j < n1; ++j)
        {
            scanf("%d", &a[(i * n1) + j]);
        }
    }
    printf("enter b\n");
    for (int i = 0; i < m2; ++i)
    {
        for (int j = 0; j < n2; ++j)
        {
            scanf("%d", &b[(i * n2) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m1 * n1);
    hipMalloc((void **)&db, sizeof(int) * m2 * n2);
    hipMalloc((void **)&dc, sizeof(int) * m1 * n2);
    hipMemcpy(da, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((n2 + TILE_WIDTH - 1) / TILE_WIDTH, (m1 + TILE_WIDTH - 1) / TILE_WIDTH);
    mul<<<dimGrid, dimBlock>>>(da, db, dc, n1, n2);
    hipMemcpy(c, dc, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < m1; ++i)
    {
        for (int j = 0; j < n2; ++j)
        {
            printf("%d ", c[(i * n2) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}