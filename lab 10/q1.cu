#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_SIZE 2

__global__ void mul(int *a, int *b, int *c, int a_m, int a_n, int b_n)
{
    __shared__ int m[TILE_SIZE][TILE_SIZE];
    __shared__ int n[TILE_SIZE][TILE_SIZE];

    int row = (blockDim.y * blockIdx.y) + threadIdx.y;
    int col = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (row < a_m && col < b_n)
    {
        // note it is a_n and not a_m
        int total_phases = (a_n + TILE_SIZE - 1) / TILE_SIZE;
        int pval = 0;
        for (int phases = 0; phases < total_phases; ++phases)
        {
            int tiled_col = (phases * TILE_SIZE) + threadIdx.x;
            int tiled_row = (phases * TILE_SIZE) + threadIdx.y;
            if (tiled_col < a_n && tiled_row < a_n)
            {
                m[threadIdx.y][threadIdx.x] = a[(row * a_n) + tiled_col];
                n[threadIdx.y][threadIdx.x] = b[(tiled_row * b_n) + col];
            }
            else
            {
                m[threadIdx.y][threadIdx.x] = 0;
                n[threadIdx.y][threadIdx.x] = 0;
            }
            __syncthreads();
            for (int k = 0; k < TILE_SIZE; ++k)
            {
                pval += m[threadIdx.y][k] * n[k][threadIdx.x];
            }
            __syncthreads();
        }
        c[(row * b_n) + col] = pval;
    }
    return;
}

int main(){
    int *a, *b, *c, m1, n1, m2, n2, *da, *db, *dc;
    printf("enter m1, n1, m2, n2\n");
    scanf("%d %d %d %d", &m1, &n1, &m2, &n2);
    a = (int *)malloc(sizeof(int) * m1 * n1);
    b = (int *)malloc(sizeof(int) * m2 * n2);
    c = (int *)malloc(sizeof(int) * m1 * n2);
    printf("enter a\n");
    for (int i = 0; i < m1; ++i)
    {
        for (int j = 0; j < n1; ++j)
        {
            scanf("%d", &a[(i * n1) + j]);
        }
    }
    printf("enter b\n");
    for (int i = 0; i < m2; ++i)
    {
        for (int j = 0; j < n2; ++j)
        {
            scanf("%d", &b[(i * n2) + j]);
        }
    }
    hipMalloc((void **)&da, sizeof(int) * m1 * n1);
    hipMalloc((void **)&db, sizeof(int) * m2 * n2);
    hipMalloc((void **)&dc, sizeof(int) * m1 * n2);
    hipMemcpy(da, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    // note order x, y
    dim3 dimGrid((n2 + TILE_SIZE - 1) / TILE_SIZE, (m1 + TILE_SIZE - 1) / TILE_SIZE);
    mul<<<dimGrid, dimBlock>>>(da, db, dc, m1, n1, n2);
    hipMemcpy(c, dc, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    printf("answer\n");
    for (int i = 0; i < m1; ++i)
    {
        for (int j = 0; j < n2; ++j)
        {
            printf("%d ", c[(i * n2) + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}